#include "hip/hip_runtime.h"
/**
 * @file main.cu (implementation file)
 *
 * @brief Problem name: CUDA matrix multiplication along rows
 *
 *
 * @author Pedro Casimiro, nmec: 93179
 * @author Diogo Bento, nmec: 93391
 */


#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <libgen.h>
#include <string.h>
#include <time.h>

/**
 * @brief We know our hardware supports 1024 concurrent threads
 * 
 */
static const int MAX_THREADS = 1024;

/**
 * @brief Struct containing the command line argument values.
 *
 * @param status if the file was called correctly
 * @param fileCount count of the files given
 * @param fileNames array of file names given
 */
typedef struct CMDArgs
{
    int status;
    int fileCount;
    char **fileNames;
} CMDArgs;

/**
 * @brief Struct containing the results calculated from a file.
 *
 * @param marixCount number of matrices in the file.
 * @param determinants array with the determinant of all matrices.
 */
typedef struct Result
{
    int matrixCount;
    double *determinants;
} Result;

/**
 * @brief Prints correct usage of this file.
 *
 * @param cmdName name of the file
 */
static void printUsage(char *cmdName)
{
    fprintf(stderr, "\nSynopsis: %s OPTIONS [filenames]\n"
                    "  OPTIONS:\n"
                    "  -h      --- print this help\n"
                    "  -f      --- file names, space separated\n",
            cmdName);
}

/**
 * @brief Processes the command line and returns a struct with the argument values.
 *
 * @param argc argument count
 * @param args argument array
 * @return CMDArgs struct with all the argument values
 */
CMDArgs parseCMD(int argc, char *args[])
{
    CMDArgs cmdArgs;
    cmdArgs.status = EXIT_FAILURE;
    int opt;
    opterr = 0;
    int filestart = -1;
    int filespan = 0;

    if (argc == 1) // no args
    {
        fprintf(stderr, "%s: invalid format\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    do
    {
        switch ((opt = getopt(argc, args, "f:w:h")))
        {
        case 'f':                // file name
            if (filestart != -1) // duplicate -f
            {
                fprintf(stderr, "%s: -f can only be used once\n", basename(args[0]));
                printUsage(basename(args[0]));
                return cmdArgs;
            }
            filestart = optind - 1;
            for (filespan = 0; filestart + filespan < argc && args[filespan + filestart][0] != '-'; filespan++)
            {
                // constantly checks if within bounds and isnt next OPT
                // this loop only serves to advance filespan
            }
            cmdArgs.fileCount = filespan;
            cmdArgs.fileNames = (char **)malloc(sizeof(char **) * filespan);
            memcpy(cmdArgs.fileNames, &args[filestart], (sizeof(char *) * filespan));
            break;
        case 'h': // help
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        case '?': // invalid option
            fprintf(stderr, "%s: invalid option\n", basename(args[0]));
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        default: // -1
            break;
        }
    } while (opt != -1);
    if (filestart == -1 || filespan == 0) // no files
    {
        fprintf(stderr, "%s: file name is missing\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    cmdArgs.status = EXIT_SUCCESS;
    return cmdArgs;
}

/**
 * @brief Prints program results.
 *
 * @param fileNames names of processed files
 * @param fileCount how many files were processed
 */
static void printResults(char **fileNames, int fileCount, Result* results)
{
    printf("%-50s %6s %30s\n", "File name", "Matrix", "Determinant");
    for (int i = 0; i < fileCount; i++)
    {
        for (int j = 0; j < results[i].matrixCount; j++)
        {
            printf("%-50s %6d %30.5e\n", fileNames[i], j + 1, results[i].determinants[j]);
        }
    }
}


// grid 1D block 1D
__global__ void calculateDeterminantsOnGPU(double *matrix, double * determinants, int order, int offset, int totalMatrices)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int localMatrixOffset = blockIdx.x*order*order; //idx/order
    unsigned short row = idx%order;
    double hold;

    //for when matrixes arent a multiple of how many we can handle at once, kill excess blocks
    if (offset+blockIdx.x>=totalMatrices){
        return;
    }
    //initialize relevant shared memory, this won't write out of bound because we've forced return on out of bound entities.
    if (row == 0)
        determinants[offset+blockIdx.x] = 1;

    
    for (short i=0;i<order;i++){
        if (matrix[localMatrixOffset+i*order+i]==0){
            
            short foundJ = 0;
            for (short j = i + 1; j < order; j++) 
                if (matrix[localMatrixOffset+ order * i + j] != 0) //this searches COLUMNS
                    foundJ = j;
                    break;
            
            if (!foundJ){ //no swap possible
                if (row==0){
                    determinants[offset+blockIdx.x]=0; //set value before exit
                }
                return;
            }

            __syncthreads(); //SYNC POINT: WE KNOW WHAT SWAP IS REQUIRED
            
            if (row>=i){
            //perform swap by grabbing value from row ROW, column FOUNDJ into row ROW column I
            hold = matrix[localMatrixOffset+row*order+foundJ];
            matrix[localMatrixOffset+row*order+foundJ] =  matrix[localMatrixOffset+row*order+i];
            matrix[localMatrixOffset+row*order+i] = hold;
            }
            __syncthreads(); //SYNC POINT: SWAPS HAVE BEEN PERFORMED
            if (row==i){
                determinants[offset+blockIdx.x]*=-1;
            }
        }
        if (row==i){
                determinants[offset+blockIdx.x]*=matrix[localMatrixOffset+i*order+i];
            }
        if (row>i){
            //REDUCE ALONG ROW
            hold = matrix[localMatrixOffset+order*row+i]/matrix[localMatrixOffset+i*order+i]; //A(k,i) /A(i,i)
            for (int j = i; j < order; j++)
            {
                matrix[localMatrixOffset+ row * order + j] -= hold * matrix[localMatrixOffset+i * order + j];
            }
        }

        __syncthreads(); //SYNC POINT: REDUCE IS DONE
            
    }


}

static void parseFile(char * fileName, Result* resultSlot){

    FILE *file = fopen(fileName, "rb");
    // if file is a dud
    if (file == NULL)
    {
        (*resultSlot).matrixCount = 0;
        return;
    }
    // number of matrices in the file
    int count;
    fread(&count, 4, 1, file);

    // order of the matrices in the file
    int order;
    fread(&order, 4, 1, file);
    if (order>MAX_THREADS){
        printf("File %s with matrixes of size %d is larger than our limit %d, it will be ignored\n",fileName,count,MAX_THREADS);
        fclose(file);
        (*resultSlot).matrixCount = 0;
        return;

    }
    
    //initialize results object
    (*resultSlot).matrixCount = count;
    (*resultSlot).determinants = (double *) malloc(sizeof(double)*count);
    double * determinantsOnGPU;
    CHECK(hipMalloc((void **)&determinantsOnGPU, sizeof(double)*count));
    //how many matrixes we can work with at once
    int simultaneousMatrixes = MAX_THREADS/order;

    int memsize = simultaneousMatrixes*order * order* sizeof(double);
    double * matrixOnGPU;
    double * matrix = (double *) malloc(memsize);
    CHECK(hipMalloc((void **)&matrixOnGPU, memsize));
    dim3 block(order, 1);
    dim3 grid((MAX_THREADS + order - 1) / order);
    
    for (int i = 0; i < count/simultaneousMatrixes; i++)
        {
            fread(matrix, 8, memsize, file);
            CHECK(hipMemcpy(matrixOnGPU, matrix, memsize, hipMemcpyHostToDevice));
            calculateDeterminantsOnGPU<<<grid, block>>>(matrixOnGPU, determinantsOnGPU, order, i*simultaneousMatrixes,count);
            CHECK(hipDeviceSynchronize());
            CHECK(hipGetLastError());
            
        }
    //copy results out of device
    CHECK(hipMemcpy((*resultSlot).determinants ,determinantsOnGPU,  count , hipMemcpyDeviceToHost));
    
    //free memory
    CHECK(hipFree(determinantsOnGPU));
    CHECK(hipFree(matrixOnGPU));
    free(matrix);
    fclose(file);
}


int main(int argc, char **argv)
{

    struct timespec start, finish; // time measurement

    CMDArgs cmdArgs = parseCMD(argc, argv);
    if (cmdArgs.status == EXIT_FAILURE)
        return EXIT_FAILURE;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    
    Result * results = (Result *) malloc( sizeof(Result)*cmdArgs.fileCount);

    clock_gettime(CLOCK_MONOTONIC_RAW, &start); // begin time measurement

    for (int i =0;i<cmdArgs.fileCount;i++){
        parseFile(cmdArgs.fileNames[i],results+i);
    }
    clock_gettime(CLOCK_MONOTONIC_RAW, &finish); // end time measurement
    printf("results\n");
    printResults(cmdArgs.fileNames,cmdArgs.fileCount,results);
    printf("\nElapsed time = %.6f s\n", (finish.tv_sec - start.tv_sec) / 1.0 + (finish.tv_nsec - start.tv_nsec) / 1000000000.0);
    
    free(cmdArgs.fileNames);
    for (int i =0;i<cmdArgs.fileCount;i++){
        if (results[i].matrixCount)
            free(results[i].determinants);
    }
    free(results);
    
    CHECK(hipDeviceReset());

    return (0);

}

