#include "hip/hip_runtime.h"
/**
 * @file main.cu (implementation file)
 *
 * @brief Problem name: CUDA matrix multiplication along rows
 *
 *
 * @author Pedro Casimiro, nmec: 93179
 * @author Diogo Bento, nmec: 93391
 */


#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <libgen.h>
#include <string.h>
#include <time.h>


/**
 * @brief Struct containing the command line argument values.
 *
 * @param status if the file was called correctly
 * @param fileCount count of the files given
 * @param fileNames array of file names given
 */
typedef struct CMDArgs
{
    int status;
    int fileCount;
    char **fileNames;
} CMDArgs;

/**
 * @brief Struct containing the results calculated from a file.
 *
 * @param marixCount number of matrices in the file.
 * @param determinants array with the determinant of all matrices.
 */
typedef struct Result
{
    int matrixCount;
    double *determinants;
} Result;

/**
 * @brief Prints correct usage of this file.
 *
 * @param cmdName name of the file
 */
static void printUsage(char *cmdName)
{
    fprintf(stderr, "\nSynopsis: %s OPTIONS [filenames]\n"
                    "  OPTIONS:\n"
                    "  -h      --- print this help\n"
                    "  -f      --- file names, space separated\n",
            cmdName);
}

/**
 * @brief Processes the command line and returns a struct with the argument values.
 *
 * @param argc argument count
 * @param args argument array
 * @return CMDArgs struct with all the argument values
 */
CMDArgs parseCMD(int argc, char *args[])
{
    CMDArgs cmdArgs;
    cmdArgs.status = EXIT_FAILURE;
    int opt;
    opterr = 0;
    int filestart = -1;
    int filespan = 0;

    if (argc == 1) // no args
    {
        fprintf(stderr, "%s: invalid format\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    do
    {
        switch ((opt = getopt(argc, args, "f:w:h")))
        {
        case 'f':                // file name
            if (filestart != -1) // duplicate -f
            {
                fprintf(stderr, "%s: -f can only be used once\n", basename(args[0]));
                printUsage(basename(args[0]));
                return cmdArgs;
            }
            filestart = optind - 1;
            for (filespan = 0; filestart + filespan < argc && args[filespan + filestart][0] != '-'; filespan++)
            {
                // constantly checks if within bounds and isnt next OPT
                // this loop only serves to advance filespan
            }
            cmdArgs.fileCount = filespan;
            cmdArgs.fileNames = (char **)malloc(sizeof(char **) * filespan);
            memcpy(cmdArgs.fileNames, &args[filestart], (sizeof(char *) * filespan));
            break;
        case 'h': // help
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        case '?': // invalid option
            fprintf(stderr, "%s: invalid option\n", basename(args[0]));
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        default: // -1
            break;
        }
    } while (opt != -1);
    if (filestart == -1 || filespan == 0) // no files
    {
        fprintf(stderr, "%s: file name is missing\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    cmdArgs.status = EXIT_SUCCESS;
    return cmdArgs;
}

/**
 * @brief Prints program results.
 *
 * @param fileNames names of processed files
 * @param fileCount how many files were processed
 */
static void printResults(char **fileNames, int fileCount, Result* results)
{
    printf("%-50s %6s %30s\n", "File name", "Matrix", "Determinant");
    for (int i = 0; i < fileCount; i++)
    {
        for (int j = 0; j < results[i].matrixCount; j++)
        {
            printf("%-50s %6d %30.5e\n", fileNames[i], j + 1, results[i].determinants[j]);
        }
    }
}


/**
 * @brief Function responsible for computing determinants on GPU
 * 
 * @param matrix pointer containing all matrixes
 * @param determinants pointer containing determinant slots
 * @param order size of matrices
 * @return
 */
__global__ void calculateDeterminantsOnGPU(double *matrix, double * determinants, int order)
{
    //matrix we are working with
    unsigned int bx = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x* gridDim.y* blockIdx.z;
    //row we are working with
    unsigned int idx = threadIdx.x+ blockDim.x* threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;

    //point at our matrix
    matrix+=bx*order*order;

    //point at our row
    //matrix+= idx*order;

    double hold;

    //initialize relevant shared memory
    if (idx == 0){
        determinants[bx] = 1;
    }
    
    for (short i=0;i<order;i++){
        if (matrix[i*order+i]==0){
            
            short foundJ = 0;
            for (short j = i + 1; j < order; j++) 
                if (matrix[order * i + j] != 0) //this searches COLUMNS
                    foundJ = j;
                    break;
            
            if (!foundJ){ //no swap possible
                if (idx==0){
                    determinants[bx]=0; //set value before exit
                }
                return;
            }

            __syncthreads(); //SYNC POINT: WE KNOW WHAT SWAP IS REQUIRED
            
            if (idx>=i){
            //perform swap by grabbing value from row ROW, column FOUNDJ into row ROW column I
            hold = matrix[idx*order + foundJ];
            matrix[idx*order + foundJ] =  matrix[idx*order + i];
            matrix[idx*order + i] = hold;
            }


            __syncthreads(); //SYNC POINT: SWAPS HAVE BEEN PERFORMED
            if (idx==i){
                determinants[bx]*=-1;
            }
        }
        if (idx==i){
                determinants[bx]*=matrix[idx*order+idx];
            }
        if (idx>i){
            //REDUCE ALONG ROW
            hold = matrix[idx*order + i]/matrix[i*order+i]; //A(k,i) /A(i,i)
            for (int j = i; j < order; j++)
            {
                matrix[idx*order + j] -= hold * matrix[i* order + j];
            }
        }

        __syncthreads(); //SYNC POINT: REDUCE IS DONE
            
    }


}

static void parseFile(char * fileName, Result* resultSlot){

    FILE *file = fopen(fileName, "rb");
    // if file is a dud
    if (file == NULL)
    {
        (*resultSlot).matrixCount = 0;
        return;
    }
    // number of matrices in the file
    int count;
    fread(&count, 4, 1, file);

    // order of the matrices in the file
    int order;
    fread(&order, 4, 1, file);

    if (order*order*count+count>(size_t) 5e9){
        printf("File %s is bigger than we can handle, it will be ignored\n",fileName);
        fclose(file);
        (*resultSlot).matrixCount = 0;
        return;

    }
    
    //initialize results object
    (*resultSlot).matrixCount = count;
    (*resultSlot).determinants = (double *) malloc(sizeof(double)*count);
    double * determinantsOnGPU;
    CHECK(hipMalloc((void **)&determinantsOnGPU, sizeof(double)*count));

    int memsize = order * order * count * sizeof(double);
    
    double * matrixOnGPU;
    double * matrix = (double *) malloc(memsize);
    CHECK(hipMalloc((void **)&matrixOnGPU, memsize));
    
    dim3 block(order, 1);
    dim3 grid(count);
    

    fread(matrix, 8, memsize, file);
    CHECK(hipMemcpy(matrixOnGPU, matrix, memsize, hipMemcpyHostToDevice));
    calculateDeterminantsOnGPU<<<grid, block>>>(matrixOnGPU, determinantsOnGPU, order);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
            
    //copy results out of device
    CHECK(hipMemcpy((*resultSlot).determinants ,determinantsOnGPU,  count*sizeof(double) , hipMemcpyDeviceToHost));
    
    //free memory
    CHECK(hipFree(determinantsOnGPU));
    CHECK(hipFree(matrixOnGPU));
    fclose(file);
    free(matrix);
}


int main(int argc, char **argv)
{

    struct timespec start, finish; // time measurement

    CMDArgs cmdArgs = parseCMD(argc, argv);
    if (cmdArgs.status == EXIT_FAILURE)
        return EXIT_FAILURE;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    
    Result * results = (Result *) malloc( sizeof(Result)*cmdArgs.fileCount);

    clock_gettime(CLOCK_MONOTONIC_RAW, &start); // begin time measurement

    for (int i =0;i<cmdArgs.fileCount;i++){
        parseFile(cmdArgs.fileNames[i],results+i);
    }
    clock_gettime(CLOCK_MONOTONIC_RAW, &finish); // end time measurement
    printResults(cmdArgs.fileNames,cmdArgs.fileCount,results);
    printf("\nElapsed time = %.6f s\n", (finish.tv_sec - start.tv_sec) / 1.0 + (finish.tv_nsec - start.tv_nsec) / 1000000000.0);
    
    free(cmdArgs.fileNames);
    for (int i =0;i<cmdArgs.fileCount;i++){
        if (results[i].matrixCount)
            free(results[i].determinants);
    }
    free(results);
    
    CHECK(hipDeviceReset());

    return (0);

}

