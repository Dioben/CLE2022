#include "hip/hip_runtime.h"
/**
 * @file main.cu (implementation file)
 *
 * @brief Problem name: CUDA matrix multiplication along columns
 *
 *
 * @author Pedro Casimiro, nmec: 93179
 * @author Diogo Bento, nmec: 93391
 */

#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <libgen.h>
#include <string.h>
#include <time.h>

/**
 * @brief Struct containing the command line argument values.
 *
 * @param status if the file was called correctly
 * @param fileCount count of the files given
 * @param fileNames array of file names given
 */
typedef struct CMDArgs
{
    int status;
    int fileCount;
    char **fileNames;
} CMDArgs;

/**
 * @brief Struct containing the results calculated from a file.
 *
 * @param marixCount number of matrices in the file.
 * @param determinants array with the determinant of all matrices.
 */
typedef struct Result
{
    int matrixCount;
    double *determinants;
} Result;

/**
 * @brief Prints correct usage of this file.
 *
 * @param cmdName name of the file
 */
static void printUsage(char *cmdName)
{
    fprintf(stderr, "\nSynopsis: %s OPTIONS [filenames]\n"
                    "  OPTIONS:\n"
                    "  -h      --- print this help\n"
                    "  -f      --- file names, space separated\n",
            cmdName);
}

/**
 * @brief Processes the command line and returns a struct with the argument values.
 *
 * @param argc argument count
 * @param args argument array
 * @return CMDArgs struct with all the argument values
 */
CMDArgs parseCMD(int argc, char *args[])
{
    CMDArgs cmdArgs;
    cmdArgs.status = EXIT_FAILURE;
    int opt;
    opterr = 0;
    int filestart = -1;
    int filespan = 0;

    if (argc == 1) // no args
    {
        fprintf(stderr, "%s: invalid format\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    do
    {
        switch ((opt = getopt(argc, args, "f:w:h")))
        {
        case 'f':                // file name
            if (filestart != -1) // duplicate -f
            {
                fprintf(stderr, "%s: -f can only be used once\n", basename(args[0]));
                printUsage(basename(args[0]));
                return cmdArgs;
            }
            filestart = optind - 1;
            for (filespan = 0; filestart + filespan < argc && args[filespan + filestart][0] != '-'; filespan++)
            {
                // constantly checks if within bounds and isnt next OPT
                // this loop only serves to advance filespan
            }
            cmdArgs.fileCount = filespan;
            cmdArgs.fileNames = (char **)malloc(sizeof(char **) * filespan);
            memcpy(cmdArgs.fileNames, &args[filestart], (sizeof(char *) * filespan));
            break;
        case 'h': // help
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        case '?': // invalid option
            fprintf(stderr, "%s: invalid option\n", basename(args[0]));
            printUsage(basename(args[0]));
            if (!(filestart == -1 || filespan == 0))
                free(cmdArgs.fileNames);
            return cmdArgs;
        default: // -1
            break;
        }
    } while (opt != -1);
    if (filestart == -1 || filespan == 0) // no files
    {
        fprintf(stderr, "%s: file name is missing\n", basename(args[0]));
        printUsage(basename(args[0]));
        return cmdArgs;
    }
    cmdArgs.status = EXIT_SUCCESS;
    return cmdArgs;
}

/**
 * @brief Prints program results.
 *
 * @param fileNames names of processed files
 * @param fileCount how many files were processed
 */
static void printResults(char **fileNames, int fileCount, Result *results)
{
    printf("%-50s %6s %30s\n", "File name", "Matrix", "Determinant");
    for (int i = 0; i < fileCount; i++)
    {
        for (int j = 0; j < results[i].matrixCount; j++)
        {
            printf("%-50s %6d %30.5e\n", fileNames[i], j + 1, results[i].determinants[j]);
        }
    }
}

/**
 * @brief Calculates the determinant of a matrix through Gaussian elimination.
 *
 * @param order order of the matrix
 * @param matrix 1D representation of the matrix
 * @return determinant of the matrix
 */
static double calculateDeterminantOnCPU(int order, double *matrix)
{
    // if matrix is small do a simpler calculation
    if (order == 1)
    {
        return matrix[0];
    }
    else if (order == 2)
    {
        return matrix[0] * matrix[3] - matrix[1] * matrix[2]; // AD - BC
    }
    double determinant = 1;
    double hold;
    // turn matrix into a triangular form
    for (int i = 0; i < order; i++)
    {
        // if diagonal is 0 swap rows with another whose value in that column is not 0
        if (matrix[i * order + i] == 0)
        {
            int foundJ = 0;
            for (int j = i + 1; j < order; j++)
                if (matrix[i * order + j] != 0)
                { // scan for column
                    foundJ = j;
                    break;
                }
            if (!foundJ)
                return 0;
            determinant *= -1;
            for (int swap = i; swap < order; swap++)
            { // swap column i, foundj
                hold = matrix[i * order + swap];
                matrix[i * order + swap] = matrix[foundJ * order + swap];
                matrix[foundJ * order + swap] = hold;
            }
        }

        // reduce matrix
        for (int j = i + 1; j < order; j++)
        {
            hold = matrix[i * order + j] / matrix[i * order + i]; //(i,j)/(i,i)
            for (int k = i + 1; k < order; k++)
            {
                matrix[k * order + j] -= hold * matrix[k * order + i];
            }
        }
        determinant *= matrix[i * order + i];
    }

    return determinant;
}

/**
 * @brief Function responsible for computing determinants on GPU
 *
 * @param matrix pointer containing all matrixes
 * @param determinants pointer containing determinant slots
 * @param order size of matrices
 * @return
 */
__global__ void calculateDeterminantsOnGPU(double *matrix, double *determinants, int order)
{
    // matrix we are working with
    unsigned int bx = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
    // column we are working with
    unsigned int idx = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;

    // point at our matrix
    matrix += bx * order * order;

    // point at our column
    double *threadcolumn = matrix + idx;

    double hold;

    // initialize relevant shared memory
    if (idx == 0)
    {
        determinants[bx] = 1;
    }

    for (short i = 0; i < order; i++)
    {
        double *itercolumn = matrix + i;
        if (itercolumn[i * order] == 0)
        {
            short foundJ = 0;
            for (short j = i + 1; j < order; j++)
            {
                if (itercolumn[j * order] != 0)
                { // this searches ROWS
                    foundJ = j;
                    break;
                }
            }
            if (!foundJ)
            { // no swap possible
                if (idx == 0)
                {
                    determinants[bx] = 0; // set value before exit
                }
                return;
            }

            __syncthreads(); // SYNC POINT: WE KNOW WHAT SWAP IS REQUIRED

            if (idx >= i)
            {
                // perform swap by grabbing value from row OTHERJ, column COLUMN into row I column COLUMN
                hold = threadcolumn[foundJ * order];
                threadcolumn[foundJ * order] = threadcolumn[i * order];
                threadcolumn[i * order] = hold;
            }

            __syncthreads(); // SYNC POINT: SWAPS HAVE BEEN PERFORMED

            if (idx == i)
            {
                determinants[bx] *= -1;
            }
        }
        if (idx == i)
        {
            determinants[bx] *= threadcolumn[idx * order];
        }
        if (idx > i)
        {
            // REDUCE ALONG COLUMN
            hold = threadcolumn[i * order] / itercolumn[i * order]; // A(i,j) /A(i,i)
            for (int k = i + 1; k < order; k++)
            {
                threadcolumn[k * order] -= hold * itercolumn[k * order];
            }
        }

        __syncthreads(); // SYNC POINT: REDUCE IS DONE
    }
}

/**
 * @brief Parses file contents and calculates determinants on GPU
 *
 * @param fileName Name of file to handle
 * @param resultSlot Results object to write to
 */
static void parseFile(char *fileName, Result *resultSlot)
{
    FILE *file = fopen(fileName, "rb");
    // if file is a dud
    if (file == NULL)
    {
        (*resultSlot).matrixCount = 0;
        return;
    }
    // number of matrices in the file
    int count;
    fread(&count, 4, 1, file);

    // order of the matrices in the file
    int order;
    fread(&order, 4, 1, file);

    if ((size_t)order * (size_t)order * (size_t)count + (size_t)count > (size_t)5e9)
    {
        printf("File %s is bigger than we can handle, it will be ignored\n", fileName);
        fclose(file);
        (*resultSlot).matrixCount = 0;
        return;
    }

    // initialize results object
    (*resultSlot).matrixCount = count;
    (*resultSlot).determinants = (double *)malloc(sizeof(double) * count);
    double *determinantsOnGPU;
    CHECK(hipMalloc((void **)&determinantsOnGPU, sizeof(double) * count));

    int memsize = order * order * count * sizeof(double);

    double *matrixOnGPU;
    double *matrix = (double *)malloc(memsize);
    CHECK(hipMalloc((void **)&matrixOnGPU, memsize));

    dim3 block(order, 1);
    dim3 grid(count);

    fread(matrix, 8, memsize, file);
    CHECK(hipMemcpy(matrixOnGPU, matrix, memsize, hipMemcpyHostToDevice));
    calculateDeterminantsOnGPU<<<grid, block>>>(matrixOnGPU, determinantsOnGPU, order);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // copy results out of device
    CHECK(hipMemcpy((*resultSlot).determinants, determinantsOnGPU, count * sizeof(double), hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(determinantsOnGPU));
    CHECK(hipFree(matrixOnGPU));
    fclose(file);
    free(matrix);
}

/**
 * @brief Parses file contents and calculates determinants on CPU
 * Used for comparison purposes
 *
 * @param fileName Name of file to handle
 * @param resultSlot Results object to write to
 */
static void parseFileOnCPU(char *fileName, Result *resultSlot)
{
    FILE *file = fopen(fileName, "rb");
    // if file is a dud
    if (file == NULL)
    {
        (*resultSlot).matrixCount = 0;
        return;
    }
    // number of matrices in the file
    int count;
    fread(&count, 4, 1, file);

    // order of the matrices in the file
    int order;
    fread(&order, 4, 1, file);

    // initialize results object
    (*resultSlot).matrixCount = count;
    (*resultSlot).determinants = (double *)malloc(sizeof(double) * count);

    double *matrix = (double *)malloc(order * order * sizeof(double));
    for (int i = 0; i < count; i++)
    {
        fread(matrix, 8, order * order, file);
        resultSlot->determinants[i] = calculateDeterminantOnCPU(order, matrix);
    }
    fclose(file);
    free(matrix);
}

/**
 * @brief Count number of different elements between 2 arrays
 *
 * @param arr1 First array
 * @param arr2 Second array
 * @param len Length of arrays
 * @param tolerance Maximum value difference
 * @return int Number of different data points
 */
static int countDifferent(double *arr1, double *arr2, int len, double tolerance)
{
    int c = 0;
    for (int i = 0; i < len; i++)
    {
        if (arr1[i] == 0)
        {
            if (fabs(arr1[i] - arr2[i]) > tolerance)
                c++;
        }
        else
        {
            if ((fabs(arr1[i] - arr2[i]) / arr1[i]) > tolerance)
                c++;
        }
    }
    return c;
}

int main(int argc, char **argv)
{
    struct timespec start, finish; // time measurement

    CMDArgs cmdArgs = parseCMD(argc, argv);
    if (cmdArgs.status == EXIT_FAILURE)
        return EXIT_FAILURE;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    Result *results = (Result *)malloc(sizeof(Result) * cmdArgs.fileCount);

    clock_gettime(CLOCK_MONOTONIC_RAW, &start); // begin time measurement

    for (int i = 0; i < cmdArgs.fileCount; i++)
    {
        parseFile(cmdArgs.fileNames[i], results + i);
    }
    clock_gettime(CLOCK_MONOTONIC_RAW, &finish); // end time measurement
    printResults(cmdArgs.fileNames, cmdArgs.fileCount, results);
    printf("\nElapsed time on GPU = %.6f s\n", (finish.tv_sec - start.tv_sec) / 1.0 + (finish.tv_nsec - start.tv_nsec) / 1000000000.0);

    Result *resultsOnCPU = (Result *)malloc(sizeof(Result) * cmdArgs.fileCount);
    clock_gettime(CLOCK_MONOTONIC_RAW, &start); // begin time measurement
    for (int i = 0; i < cmdArgs.fileCount; i++)
    {
        parseFileOnCPU(cmdArgs.fileNames[i], resultsOnCPU + i);
    }
    clock_gettime(CLOCK_MONOTONIC_RAW, &finish); // end time measurement
    printf("Elapsed time on CPU = %.6f s\n", (finish.tv_sec - start.tv_sec) / 1.0 + (finish.tv_nsec - start.tv_nsec) / 1000000000.0);

    int totaldiff = 0;
    for (int i = 0; i < cmdArgs.fileCount; i++)
    {
        int diff = 0;
        diff = countDifferent(results[i].determinants, resultsOnCPU[i].determinants, results[i].matrixCount, 5e-7);
        totaldiff += diff;
        if (diff)
            printf("Spotted %d different results at file %s\n", diff, cmdArgs.fileNames[i]);
    }
    if (!totaldiff)
        printf("\nAll values are the same between CPU and GPU\n");
    free(cmdArgs.fileNames);
    for (int i = 0; i < cmdArgs.fileCount; i++)
    {
        if (results[i].matrixCount)
            free(results[i].determinants);
        if (resultsOnCPU[i].matrixCount)
            free(resultsOnCPU[i].determinants);
    }
    free(results);
    free(resultsOnCPU);

    CHECK(hipDeviceReset());

    return (0);
}
